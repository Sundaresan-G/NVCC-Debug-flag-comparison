#include <hip/hip_runtime.h>
#include <iostream>

/*
Kernel execution time: 1.1528 ms for 1024 * 1024 * 128 float elements
*/

// Error checking macro
#define CHECK_CUDA_ERROR(call)                                                 \
    {                                                                          \
        const hipError_t error = call;                                        \
        if (error != hipSuccess) {                                            \
            std::cerr << "Error: " << __FILE__ << ":" << __LINE__ << ", "      \
                      << "code: " << error << ", reason: "                     \
                      << hipGetErrorString(error) << std::endl;               \
            exit(1);                                                           \
        }                                                                      \
    }

// Kernel for element-wise addition of float4 vectors
__global__ void addFloat4(const float4* a, const float4* b, float4* c, size_t n) {
    size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < n) {
        c[tid].x = a[tid].x + b[tid].x;
        c[tid].y = a[tid].y + b[tid].y;
        c[tid].z = a[tid].z + b[tid].z;
        c[tid].w = a[tid].w + b[tid].w;
    }
}

// Function to initialize float4 arrays
void initializeFloat4(float4* vec, size_t n) {
    for (size_t i = 0; i < n; ++i) {
        vec[i] = {1.0f, 2.0f, 3.0f, 4.0f};
    }
}

int main() {
    size_t numElements = 1024 * 1024 * 128/4;
    size_t size = numElements * sizeof(float4);

    // Allocate host memory
    float4* h_a = (float4*)malloc(size);
    float4* h_b = (float4*)malloc(size);
    float4* h_c = (float4*)malloc(size);

    if (h_a == nullptr || h_b == nullptr || h_c == nullptr) {
        std::cerr << "Failed to allocate host vectors." << std::endl;
        exit(1);
    }

    // Initialize host memory
    initializeFloat4(h_a, numElements);
    initializeFloat4(h_b, numElements);

    // Allocate device memory
    float4* d_a;
    float4* d_b;
    float4* d_c;
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_a, size));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_b, size));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_c, size));

    // Copy host memory to device
    CHECK_CUDA_ERROR(hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice));

    // Create CUDA events for timing
    hipEvent_t start, stop;
    CHECK_CUDA_ERROR(hipEventCreate(&start));
    CHECK_CUDA_ERROR(hipEventCreate(&stop));

    // Launch the kernel and measure time
    dim3 blockSize(256);
    dim3 gridSize((numElements + blockSize.x - 1) / blockSize.x);

    addFloat4<<<gridSize, blockSize>>>(d_a, d_b, d_c, numElements); //warmup

    CHECK_CUDA_ERROR(hipEventRecord(start, 0));
    addFloat4<<<gridSize, blockSize>>>(d_a, d_b, d_c, numElements);
    CHECK_CUDA_ERROR(hipEventRecord(stop, 0));
    CHECK_CUDA_ERROR(hipEventSynchronize(stop));

    // Calculate elapsed time
    float milliseconds = 0;
    CHECK_CUDA_ERROR(hipEventElapsedTime(&milliseconds, start, stop));

    // Copy result back to host
    CHECK_CUDA_ERROR(hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost));

    // Print some of the results and the execution time
    // for (size_t i = 0; i < 10; ++i) {
    //     std::cout << "h_c[" << i << "] = { "
    //               << h_c[i].x << ", " << h_c[i].y << ", " << h_c[i].z << ", "
    //               << h_c[i].w << " }" << std::endl;
    // }
    std::cout << "Kernel execution time: " << milliseconds << " ms" << std::endl;

    // Free device memory
    CHECK_CUDA_ERROR(hipFree(d_a));
    CHECK_CUDA_ERROR(hipFree(d_b));
    CHECK_CUDA_ERROR(hipFree(d_c));

    // Free host memory
    free(h_a);
    free(h_b);
    free(h_c);

    // Destroy CUDA events
    CHECK_CUDA_ERROR(hipEventDestroy(start));
    CHECK_CUDA_ERROR(hipEventDestroy(stop));

    return 0;
}